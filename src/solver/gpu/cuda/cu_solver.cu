#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>
//#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>

#include "solver/gpu/cuda/cu_solver.h"

//#include "util/cudautil.h"

//using namespace telef::solver::utils;

using Clock=std::chrono::high_resolution_clock;

#define BLOCKSIZE 128


__global__
void _print_arr(const float *arr_d, const int n) {
    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid


    // grid-striding loop
    for (int i = start_index; i < n; i += stride) {

        printf("Element[%d]: %.5f\n", i, arr_d[i]);
//        arr_d[i] += 1;
    }
}

void print_array(const char* msg, const float *arr_d, const int n) {
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((n + BLOCKSIZE - 1) / BLOCKSIZE);

    printf("%s:\n", msg);
    _print_arr << < dimGrid, dimBlock >> > (arr_d, n);
    hipDeviceSynchronize();
    printf("\n");
}

__inline__ __device__
float warpReduceSum(float val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__inline__ __device__
float blockReduceSum(float val) {

    static __shared__ float shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

    if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}

__global__
void _calc_error(float* error, const float* residuals, const int nRes){
    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    float sum = 0;
    // grid-striding loop
    for (int i = start_index; i < nRes; i += stride) {
//        printf("Thread[%d]::add[%d]: %.2f\n",threadIdx.x, i, residuals[i]);
        sum += residuals[i]*residuals[i];
    }

    sum = blockReduceSum(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd(error, sum);
    }
}

void calc_error(float* error, const float* residuals, const int nRes){
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((nRes + BLOCKSIZE - 1) / BLOCKSIZE);

    _calc_error << < dimGrid, dimBlock >> >(error, residuals, nRes);
    hipDeviceSynchronize();
}

__global__
void _cuda_step_down(float* step, float* lambda, const float* factor){
//    printf("update lambda: %.3f * %.3f = ", lambda[0], factor[0]);
    lambda[0] *= factor[0];
//    printf("%.3f\n", lambda[0]);

    step[0] = 1 + lambda[0];
//    printf("step down: 1 + %.3f = %.3f\n", lambda[0], step[0]);
}

/* stepdown (lambda*down)
* step = 1 + lambda;
*/
void cuda_step_down(float* step, float* lambda, const float* factor){
    _cuda_step_down << < 1, 1 >> >(step, lambda, factor);
    hipDeviceSynchronize();
}

/*
 * step = (1 + lambda * up) / (1 + lambda);
 * stepup (lambda*up)
 */
__global__
void _cuda_step_update(float* lambda, const float* factor){
    lambda[0] *= factor[0];
}

void cuda_step_update(float* lambda, const float* factor){
    _cuda_step_update << < 1, 1 >> >(lambda, factor);
    hipDeviceSynchronize();
}

__global__
void _update_hessians(float *hessians, float *dampeningFactors, float *lambda, int nParams, bool goodStep) {
    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    printf("_update_hessians:in");

    // grid-striding loop
    for (int i = start_index; i < nParams; i += stride) {
        int diagonal_index = i+nParams*i;
        // Apply step down diagonal
        //hessians[i+nParams*i] += hessians[i+nParams*i] * step[0];
        if (goodStep)
        {
            hessians[diagonal_index] -= dampeningFactors[i] * lambda[0] / 10.;
        }

        // adaptive scaling
        dampeningFactors[i]
                = std::max(dampeningFactors[i], hessians[diagonal_index]);

        // continuous scaling
        //scaling_vector[parameter_index] = hessian[diagonal_index];

        // initial scaling
        //if (scaling_vector[parameter_index] == 0.)
        //    scaling_vector[parameter_index] = hessian[diagonal_index];

        hessians[diagonal_index] += dampeningFactors[i] * lambda[0];

        printf("_update_hessians:hessians[%d][%d]: %.4f\n",i, i, hessians[diagonal_index]);
    }
}

void update_hessians(float *hessians, float *dampeningFactors, float *lambda, int nParams, bool goodStep) {
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((nParams + BLOCKSIZE - 1) / BLOCKSIZE);

    printf("update_hessians:block[%d] grid[%d] n: %.d\n",BLOCKSIZE, ((nParams + BLOCKSIZE - 1) / BLOCKSIZE), nParams);
    _update_hessians << < dimGrid, dimBlock >> >(hessians, dampeningFactors, lambda, nParams, goodStep);
    hipDeviceSynchronize();
    print_array("New Hessian",hessians, nParams*nParams);

}

__global__
void _update_parameters(float* newParams, const float* params, const float* newDelta, const int nParams){
    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    // grid-striding loop
    for (int i = start_index; i < nParams; i += stride) {
        // Apply step down diagonal
        newParams[i] = params[i] + newDelta[i];
//        printf("params[%d]: %.4f\n",i, newParams[i]);
    }
}

void update_parameters(float* newParams, const float* params, const float* newDelta, const int nParams){
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((nParams + BLOCKSIZE - 1) / BLOCKSIZE);

    _update_parameters << < dimGrid, dimBlock >> >(newParams, params, newDelta, nParams);
    hipDeviceSynchronize();
    //print_array("New Params", newParams, nParams);
}

void initializeSolverBuffer(hipsolverHandle_t solver_handle,
        float **solverBuffer, int &solverBufferSize, hipblasFillMode_t uplo,
        float *matrix, const int &nRows, const int &nCols) {

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

    // Allocate working space for decomposition
    // TODO: Try Allocate working space once in parameterBlock, or every-time hessian is computed
    //       as the matrix should not change much
    cusolver_status =
            hipsolverDnSpotrf_bufferSize(solver_handle, uplo,
                                        nRows, matrix, nCols,
                                        &solverBufferSize);

    // Should not happen, if it does bad stuff man...
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    hipMalloc((void**)solverBuffer, solverBufferSize * sizeof(float));
}

/**
 * This function takes a symmetric, positive-definite matrix "matA" and overwrites
 * the the lower half of "matA" with the lower-triangular Cholesky factor l for A = L * LH form.
 * Elements above the diagonal of "matA" are neither used nor modified. The decomposition is performed in place.
 *
 * @param solver_handle
 * @param cublas_handle
 * @param matA, matrix of size nxn
 * @param n, size of nxn matrix "matA"
 * @return true if matrix is positive-definite, otherwise false
 */
bool decompose_cholesky(hipsolverHandle_t solver_handle, float* matA, const int n ){

//    auto allt1 = Clock::now();
    bool decomp_status = true;
    int lda = n;
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;


    int *info_d = NULL; // info in gpu (device copy)
//    CUDA_MALLOC(&info_d, static_cast<size_t>(1));
    hipMalloc((void**)&info_d, sizeof(int));

    // Allocate working space for decomposition
    // TODO: Try Lazily initialize allocate working space once in parameterBlock, or every-time hessian is computed
    //       as the matrix should not change much?

    int buffer_size = 0;
    float *buffer_d;
    initializeSolverBuffer(solver_handle, &buffer_d, buffer_size, uplo,
                           matA, n, lda);

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    print_array("decompose_cholesky:Hessian:before", matA, n);
    // Compute A = L*LH, result in matA in lower triangular form
//    auto t1 = Clock::now();
    cusolver_status =
            hipsolverDnSpotrf(solver_handle, uplo,
                             n, matA, lda,
                             buffer_d, buffer_size,
                             info_d );
    print_array("decompose_cholesky:Decomposed Hessian:after", matA, n);
//    print_array("L",matA,n*n);
//    hipDeviceSynchronize();
//    auto t2 = Clock::now();


    if (HIPSOLVER_STATUS_SUCCESS != cusolver_status) {
        printf("hipsolverDnSpotrf failed: status %d", cusolver_status);
        decomp_status = false;
    }

    int info_h;
//    CUDA_CHECK(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost);

    if ( 0 != info_h ){
        fprintf(stderr, "Error: Cholesky factorization failed\n");
        if ( 0 > info_h ){
            printf("%d-th parameter is wrong \n", -info_h);
        }
        decomp_status = false;
    }
//    auto allt2 = Clock::now();
//    std::cout << "hipsolverDnSpotrf Time: "
//              << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()
//              << " nanoseconds" << std::endl;
//    std::cout << "Total Time: "
//              << std::chrono::duration_cast<std::chrono::nanoseconds>(allt2 - allt1).count()
//              << " nanoseconds" << std::endl;

    // free resources
    if (info_d) hipFree(info_d);
    if (buffer_d ) hipFree(buffer_d);

    return decomp_status;
}


void solve_system_cholesky(hipsolverHandle_t solver_handle, float* matA, float* matB, int n){
    int lda = n;
    int nCols_B = 1;

    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    int *info_d = NULL; // info in gpu (device copy)
//    CUDA_MALLOC(&info_d, static_cast<size_t>(1));
    hipMalloc((void**)&info_d, sizeof(int));

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

    // Compute A = L*LH, result in matA in lower triangular form
    cusolver_status =
            hipsolverDnSpotrs(solver_handle, uplo,
                             n, nCols_B, matA, lda, matB, n,
                             info_d );

//    print_array("Deltas",matB,n);

    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    int info_h;
//    CUDA_CHECK(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost);

    if ( 0 != info_h ){
        fprintf(stderr, "Error: Cholesky Solver failed\n");
        if ( 0 > info_h ){
            printf("%d-th parameter is wrong \n", -info_h);
        }
    }

    // free resources
    if (info_d) hipFree(info_d);
}